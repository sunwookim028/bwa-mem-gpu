#include "streams.cuh"
#include "errHandler.cuh"
#include "CUDAKernel_memmgnt.cuh"
#include "batch_config.h"


/* transfer index data */
static void transferIndex(
	const bwt_t *bwt, 
	const bntseq_t *bns, 
	const uint8_t *pac,
	const kmers_bucket_t *kmerHashTab,
	process_data_t *process_instance)
{
		/* CUDA GLOBAL MEMORY ALLOCATION AND TRANSFER */
	unsigned long long total_size = bwt->bwt_size*sizeof(uint32_t) + bwt->n_sa*sizeof(bwtint_t) + bns->n_seqs*sizeof(bntann1_t) + bns->n_holes*sizeof(bntamb1_t) + bns->l_pac*sizeof(uint8_t);
	fprintf(stderr, "[M::%-25s] Device memory for Index ...... %.2f MB \n", __func__, (float)total_size/MB_SIZE);

	// Burrows-Wheeler Transform
		// 1. bwt_t structure
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bwt .......... %.2f MB\n", __func__, (float)sizeof(bwt_t)/MB_SIZE);
	bwt_t* d_bwt;
	hipMalloc((void**)&d_bwt, sizeof(bwt_t));
	hipMemcpy(d_bwt, bwt, sizeof(bwt_t), hipMemcpyHostToDevice);
		// 2. int array of bwt
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bwt_int ...... %.2f MB\n", __func__, (float)bwt->bwt_size*sizeof(uint32_t)/MB_SIZE);
	uint32_t* d_bwt_int ;
	hipMalloc((void**)&d_bwt_int, bwt->bwt_size*sizeof(uint32_t));
	hipMemcpy(d_bwt_int, bwt->bwt, bwt->bwt_size*sizeof(uint32_t), hipMemcpyHostToDevice);
		// 3. int array of Suffix Array
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** suffix array . %.2f MB \n", __func__, (float)bwt->n_sa*sizeof(bwtint_t)/MB_SIZE);
	bwtint_t* d_bwt_sa ;
	hipMalloc((void**)&d_bwt_sa, bwt->n_sa*sizeof(bwtint_t));
	hipMemcpy(d_bwt_sa, bwt->sa, bwt->n_sa*sizeof(bwtint_t), hipMemcpyHostToDevice);
		// set pointers on device's memory to bwt_int and SA
	hipMemcpy((void**)&(d_bwt->bwt), &d_bwt_int, sizeof(uint32_t*), hipMemcpyHostToDevice);
	hipMemcpy((void**)&(d_bwt->sa), &d_bwt_sa, sizeof(bwtint_t*), hipMemcpyHostToDevice);

	// BNS
	// First create h_bns as a copy of bns on host
	// Then allocate its member pointers on device and copy data over
	// Then copy h_bns to d_bns
	uint32_t i, size;			// loop index and length of strings
	bntseq_t* h_bns;			// host copy to modify pointers
	h_bns = (bntseq_t*)malloc(sizeof(bntseq_t));
	memcpy(h_bns, bns, sizeof(bntseq_t));
	h_bns->anns = (bntann1_t*)malloc(bns->n_seqs*sizeof(bntann1_t));
	memcpy(h_bns->ambs, bns->ambs, bns->n_holes*sizeof(bntamb1_t));
	h_bns->ambs = (bntamb1_t*)malloc(bns->n_holes*sizeof(bntamb1_t));
	memcpy(h_bns->anns, bns->anns, bns->n_seqs*sizeof(bntann1_t));

		// allocate anns.name
	for (i=0; i<bns->n_seqs; i++){
		size = strlen(bns->anns[i].name);
		// allocate this name and copy to device
		hipMalloc((void**)&(h_bns->anns[i].name), size+1); 			// +1 for "\0"
		hipMemcpy(h_bns->anns[i].name, bns->anns[i].name, size+1, hipMemcpyHostToDevice);
	}
	// allocate anns.anno
	for (i=0; i<bns->n_seqs; i++){
		size = strlen(bns->anns[i].anno);
		// allocate this name and copy to device
		hipMalloc((void**)&(h_bns->anns[i].anno), size+1); 			// +1 for "\0"
		hipMemcpy(h_bns->anns[i].anno, bns->anns[i].anno, size+1, hipMemcpyHostToDevice);
	}
		// now h_bns->anns has pointers of name and anno on device
		// allocate anns on device and copy data from h_bns->anns to device
	bntann1_t* temp_d_anns;
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bns.anns ..... %.2f MB\n", __func__, (float)bns->n_seqs*sizeof(bntann1_t)/MB_SIZE);
	hipMalloc((void**)&temp_d_anns, bns->n_seqs*sizeof(bntann1_t));
	hipMemcpy(temp_d_anns, h_bns->anns, bns->n_seqs*sizeof(bntann1_t), hipMemcpyHostToDevice);
		// now assign this pointer to h_bns->anns
	h_bns->anns = temp_d_anns;

		// allocate bns->ambs on device and copy data to device
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bns.ambs ..... %.2f MB\n", __func__, (float)bns->n_holes*sizeof(bntamb1_t)/MB_SIZE);
	hipMalloc((void**)&h_bns->ambs, bns->n_holes*sizeof(bntamb1_t));
	hipMemcpy(h_bns->ambs, bns->ambs, bns->n_holes*sizeof(bntamb1_t), hipMemcpyHostToDevice);

		// finally allocate d_bns and copy from h_bns
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** bns .......... %.2f MB\n", __func__, (float)sizeof(bntseq_t)/MB_SIZE);
	bntseq_t* d_bns;
	hipMalloc((void**)&d_bns, sizeof(bntseq_t));
	hipMemcpy(d_bns, h_bns, sizeof(bntseq_t), hipMemcpyHostToDevice);

	// PAC
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** pac .......... %.2f MB\n", __func__, (float)bns->l_pac*sizeof(uint8_t)/MB_SIZE);
	uint8_t* d_pac ;
	hipMalloc((void**)&d_pac, bns->l_pac/4*sizeof(uint8_t)); 		// l_pac is length of ref seq
	hipMemcpy(d_pac, pac, bns->l_pac/4*sizeof(uint8_t), hipMemcpyHostToDevice); 		// divide by 4 because 2-bit encoding

	// K-MER HASH TABLE
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** kmer ......... %.2f MB\n", __func__, (float)pow4(KMER_K)*sizeof(kmers_bucket_t)/MB_SIZE);
	kmers_bucket_t* d_kmerHashTab ;
	hipMalloc((void**)&d_kmerHashTab, pow4(KMER_K)*sizeof(kmers_bucket_t)); 		// l_pac is length of ref seq
	hipMemcpy(d_kmerHashTab, kmerHashTab, pow4(KMER_K)*sizeof(kmers_bucket_t), hipMemcpyHostToDevice); 		// divide by 4 because 2-bit encoding


	// output
	process_instance->d_bwt = d_bwt;
	process_instance->d_bns = d_bns;
	process_instance->d_pac = d_pac;
	process_instance->d_kmerHashTab = d_kmerHashTab;
}

/* transfer user-defined optinos */
static void transferOptions(
	const mem_opt_t *opt, 
	mem_pestat_t *pes0,
	process_data_t *process_instance)
{
	// matching and mapping options (opt)
	mem_opt_t* d_opt;
	hipMalloc((void**)&d_opt, sizeof(mem_opt_t));
	hipMemcpy(d_opt, opt, sizeof(mem_opt_t), hipMemcpyHostToDevice);

	// paired-end stats: only allocate on device
	mem_pestat_t* d_pes;
	if (opt->flag&MEM_F_PE){
		fprintf(stderr, "[M::%-25s] pestat ....... %.2f MB\n", __func__, (float)4*sizeof(mem_pestat_t)/MB_SIZE);
		hipMalloc((void**)&d_pes, 4*sizeof(mem_pestat_t));
	}

	// output
	process_instance->d_opt = d_opt;
	process_instance->d_pes = d_pes;
	process_instance->h_pes0 = pes0;
}


/* allocate memory for intermediate data on GPU
	send pointer to process_instance
 */
void allocateIntermediateData(process_data_t *process_instance){
	unsigned long long total_size = SEQ_MAX_COUNT*sizeof(smem_aux_t) + SEQ_MAX_COUNT*sizeof(mem_seed_v) + SEQ_MAX_COUNT*sizeof(mem_chain_v) + SEQ_MAX_COUNT*500*sizeof(seed_record_t) + SEQ_MAX_COUNT*sizeof(mem_alnreg_v) + SEQ_MAX_COUNT*sizeof(mem_aln_v) + 4*5*SEQ_MAX_COUNT*sizeof(int);
	fprintf(stderr, "[M::%-25s] total intermediate data ..... %.2f MB\n", __func__, (float)total_size/MB_SIZE);
	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** aux intervals ..... %ld MB\n", __func__, SEQ_MAX_COUNT*sizeof(smem_aux_t)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&(process_instance->d_aux), SEQ_MAX_COUNT*sizeof(smem_aux_t)) );

	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** seeds array  ...... %ld MB\n", __func__, SEQ_MAX_COUNT*sizeof(mem_seed_v)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&(process_instance->d_seq_seeds), SEQ_MAX_COUNT*sizeof(mem_seed_v)) );

	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** chains ............ %ld MB\n", __func__, SEQ_MAX_COUNT*sizeof(mem_chain_v)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&(process_instance->d_chains), SEQ_MAX_COUNT*sizeof(mem_chain_v)) );

	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** seed records ...... %ld MB\n", __func__, SEQ_MAX_COUNT*500*sizeof(seed_record_t)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&(process_instance->d_seed_records), SEQ_MAX_COUNT*500*sizeof(seed_record_t)) );	// allocate enough for all seeds

	gpuErrchk( hipMalloc((void**)&(process_instance->d_Nseeds), sizeof(int)) );

	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** alignment regs .... %ld MB\n", __func__, SEQ_MAX_COUNT*sizeof(mem_alnreg_v)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&(process_instance->d_regs), SEQ_MAX_COUNT*sizeof(mem_alnreg_v)) );

	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** alignments ...... %ld MB\n", __func__, SEQ_MAX_COUNT*sizeof(mem_aln_v)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&(process_instance->d_alns), SEQ_MAX_COUNT*sizeof(mem_aln_v)) );

	if (bwa_verbose>=3) fprintf(stderr, "[M::%-25s] *** sorting keys .... %ld MB\n", __func__, 4*5*SEQ_MAX_COUNT*sizeof(int)/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&process_instance->d_sortkeys_in, SEQ_MAX_COUNT*5*sizeof(int)) );
	gpuErrchk( hipMalloc((void**)&process_instance->d_sortkeys_out, SEQ_MAX_COUNT*5*sizeof(int)) );
	gpuErrchk( hipMalloc((void**)&process_instance->d_seqIDs_in, SEQ_MAX_COUNT*5*sizeof(int)) );
	gpuErrchk( hipMalloc((void**)&process_instance->d_seqIDs_out, SEQ_MAX_COUNT*5*sizeof(int)) );
}

process_data_t* newProcess(
	const mem_opt_t *opt, 
	mem_pestat_t *pes0,
	const bwt_t *bwt, 
	const bntseq_t *bns, 
	const uint8_t *pac,
	const kmers_bucket_t *kmerHashTab
)
{
    // new instance in memory
    process_data_t *instance = (process_data_t*)calloc(1, sizeof(process_data_t));

	// user-defined options
	transferOptions(opt, pes0, instance);
    
	// transfer index data
	transferIndex(bwt, bns, pac, kmerHashTab, instance);

	// init memory management
	instance->d_buffer_pools = CUDA_BufferInit();

	// initialize intermediate processing memory on device
	allocateIntermediateData(instance);

	// initialize pinned memory for reads on host
	gpuErrchk( hipHostMalloc((void**)&instance->h_seqs, SEQ_MAX_COUNT*sizeof(bseq1_t)) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_name_ptr, SEQ_NAME_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_comment_ptr, SEQ_COMMENT_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_seq_ptr, SEQ_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_qual_ptr, SEQ_QUAL_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_sam_ptr, SEQ_SAM_LIMIT) );

	// initialize memory for reads on device
	unsigned long long total_size = SEQ_MAX_COUNT*sizeof(bseq1_t) + SEQ_NAME_LIMIT + SEQ_COMMENT_LIMIT + SEQ_LIMIT + SEQ_QUAL_LIMIT + SEQ_SAM_LIMIT;
	fprintf(stderr, "[M::%-25s] d_seqs (process) . %llu MB\n", __func__, total_size/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&instance->d_seqs, SEQ_MAX_COUNT*sizeof(bseq1_t)) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_name_ptr, SEQ_NAME_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_comment_ptr, SEQ_COMMENT_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_seq_ptr, SEQ_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_qual_ptr, SEQ_QUAL_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_sam_ptr, SEQ_SAM_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_sam_size, sizeof(int)) );

	// initialize a cuda stream for processing
	instance->CUDA_stream = malloc(sizeof(hipStream_t));
	hipStreamCreate((hipStream_t*)instance->CUDA_stream);

    return instance;
}


transfer_data_t* newTransfer(){
    transfer_data_t *instance = (transfer_data_t*)calloc(1, sizeof(transfer_data_t));

	// initialize pinned memory for reads on host
	gpuErrchk( hipHostMalloc((void**)&instance->h_seqs, SEQ_MAX_COUNT*sizeof(bseq1_t)) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_name_ptr, SEQ_NAME_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_comment_ptr, SEQ_COMMENT_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_seq_ptr, SEQ_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_qual_ptr, SEQ_QUAL_LIMIT) );
	gpuErrchk( hipHostMalloc((void**)&instance->h_seq_sam_ptr, SEQ_SAM_LIMIT) );

	// initialize memory for reads on device
	unsigned long long total_size = SEQ_MAX_COUNT*sizeof(bseq1_t) + SEQ_NAME_LIMIT + SEQ_COMMENT_LIMIT + SEQ_LIMIT + SEQ_QUAL_LIMIT + SEQ_SAM_LIMIT;
	fprintf(stderr, "[M::%-25s] d_seqs (transf) .. %llu MB\n", __func__, total_size/MB_SIZE);
	gpuErrchk( hipMalloc((void**)&instance->d_seqs, SEQ_MAX_COUNT*sizeof(bseq1_t)) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_name_ptr, SEQ_NAME_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_comment_ptr, SEQ_COMMENT_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_seq_ptr, SEQ_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_qual_ptr, SEQ_QUAL_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_sam_ptr, SEQ_SAM_LIMIT) );
	gpuErrchk( hipMalloc((void**)&instance->d_seq_sam_size, sizeof(int)) );

	// initialize a cuda stream for transfer
	instance->CUDA_stream = malloc(sizeof(hipStream_t));
	hipStreamCreate((hipStream_t*)instance->CUDA_stream);

    return instance;
}



void swapData(process_data_t *process_data, transfer_data_t *transfer_data){
	// swap host pointers
	{ auto tmp = process_data->h_seqs; process_data->h_seqs = transfer_data->h_seqs; transfer_data->h_seqs = tmp; }
	{ auto tmp = process_data->h_seq_name_ptr; process_data->h_seq_name_ptr = transfer_data->h_seq_name_ptr; transfer_data->h_seq_name_ptr = tmp; }
	{ auto tmp = process_data->h_seq_comment_ptr; process_data->h_seq_comment_ptr = transfer_data->h_seq_comment_ptr; transfer_data->h_seq_comment_ptr = tmp; }
	{ auto tmp = process_data->h_seq_seq_ptr; process_data->h_seq_seq_ptr = transfer_data->h_seq_seq_ptr; transfer_data->h_seq_seq_ptr = tmp; }
	{ auto tmp = process_data->h_seq_qual_ptr; process_data->h_seq_qual_ptr = transfer_data->h_seq_qual_ptr; transfer_data->h_seq_qual_ptr = tmp; }
	{ auto tmp = process_data->h_seq_sam_ptr; process_data->h_seq_sam_ptr = transfer_data->h_seq_sam_ptr; transfer_data->h_seq_sam_ptr = tmp; }
	// swap device pointers
	{ auto tmp = process_data->d_seqs; process_data->d_seqs = transfer_data->d_seqs; transfer_data->d_seqs = tmp; }
	{ auto tmp = process_data->d_seq_name_ptr; process_data->d_seq_name_ptr = transfer_data->d_seq_name_ptr; transfer_data->d_seq_name_ptr = tmp; }
	{ auto tmp = process_data->d_seq_comment_ptr; process_data->d_seq_comment_ptr = transfer_data->d_seq_comment_ptr; transfer_data->d_seq_comment_ptr = tmp; }
	{ auto tmp = process_data->d_seq_seq_ptr; process_data->d_seq_seq_ptr = transfer_data->d_seq_seq_ptr; transfer_data->d_seq_seq_ptr = tmp; }
	{ auto tmp = process_data->d_seq_qual_ptr; process_data->d_seq_qual_ptr = transfer_data->d_seq_qual_ptr; transfer_data->d_seq_qual_ptr = tmp; }
	{ auto tmp = process_data->d_seq_sam_ptr; process_data->d_seq_sam_ptr = transfer_data->d_seq_sam_ptr; transfer_data->d_seq_sam_ptr = tmp; }
		// swap pointer to sam_size
	{ auto tmp = process_data->d_seq_sam_size; process_data->d_seq_sam_size = transfer_data->d_seq_sam_size; transfer_data->d_seq_sam_size = tmp; }
	// swap n_seqs
	{ auto tmp = process_data->n_seqs; process_data->n_seqs = transfer_data->n_seqs; transfer_data->n_seqs = tmp; }
    return;
}

void CUDATransferSeqsIn(transfer_data_t *transfer_data){
	hipStream_t *transfer_stream = (hipStream_t*)(transfer_data->CUDA_stream);
	// copy seqs to device
	gpuErrchk( hipMemcpyAsync(transfer_data->d_seqs, transfer_data->h_seqs, transfer_data->n_seqs*sizeof(bseq1_t), hipMemcpyHostToDevice, *transfer_stream) );
	// copy name, seq, comment, qual to device
	gpuErrchk( hipMemcpyAsync(transfer_data->d_seq_name_ptr, transfer_data->h_seq_name_ptr, transfer_data->h_seq_name_size, hipMemcpyHostToDevice, *transfer_stream) );
	gpuErrchk( hipMemcpyAsync(transfer_data->d_seq_seq_ptr, transfer_data->h_seq_seq_ptr, transfer_data->h_seq_seq_size, hipMemcpyHostToDevice, *transfer_stream) );
	gpuErrchk( hipMemcpyAsync(transfer_data->d_seq_comment_ptr, transfer_data->h_seq_comment_ptr, transfer_data->h_seq_comment_size, hipMemcpyHostToDevice, *transfer_stream) );
	gpuErrchk( hipMemcpyAsync(transfer_data->d_seq_qual_ptr, transfer_data->h_seq_qual_ptr, transfer_data->h_seq_qual_size, hipMemcpyHostToDevice, *transfer_stream) );

	gpuErrchk( hipStreamSynchronize(*transfer_stream) );
}

/* copy sam output to host */
void CUDATransferSamOut(transfer_data_t *transfer_data){
	hipStream_t *transfer_stream = (hipStream_t*)(transfer_data->CUDA_stream);
	gpuErrchk( hipMemcpyAsync(transfer_data->h_seqs, transfer_data->d_seqs, transfer_data->n_seqs*sizeof(bseq1_t), hipMemcpyDeviceToHost, *transfer_stream) );
	// transfer all SAM from device to host
		// first find the total size of all SAM's
		int sam_size;
		gpuErrchk( hipMemcpyAsync(&sam_size, transfer_data->d_seq_sam_size, sizeof(int), hipMemcpyDeviceToHost, *transfer_stream) );
		// now copy
		gpuErrchk( hipMemcpyAsync(transfer_data->h_seq_sam_ptr, transfer_data->d_seq_sam_ptr, sam_size, hipMemcpyDeviceToHost, *transfer_stream) );

	hipStreamSynchronize(*transfer_stream);

	// after GPU processing, seqs[i].sam are offset. Now we need to convert this offset to actual location
	bseq1_t *seqs = transfer_data->h_seqs;
	char *sam = transfer_data->h_seq_sam_ptr;
	for (int i=0; i<transfer_data->n_seqs; i++)
		seqs[i].sam = sam + (long)seqs[i].sam;
}



void resetProcess(process_data_t *process_data){
	hipStream_t *process_stream = (hipStream_t*)(process_data->CUDA_stream);
	// reset memory management
	CUDAResetBufferPool(process_data->d_buffer_pools, *process_stream);

	// reset intermediate data
		// reset seeds
	gpuErrchk( hipMemsetAsync(process_data->d_Nseeds, 0, sizeof(int), *process_stream) );
		// reset sam size on device
	gpuErrchk( hipMemsetAsync(process_data->d_seq_sam_size, 0, sizeof(int), *process_stream) );

}

void resetTransfer(transfer_data_t *transfer_data){
	// reset name, seq, comment, qual sizes
	transfer_data->h_seq_name_size = 0;
	transfer_data->h_seq_seq_size = 0;
	transfer_data->h_seq_comment_size = 0;
	transfer_data->h_seq_qual_size = 0;
	// reset n_seqs
	transfer_data->n_seqs = 0;
}